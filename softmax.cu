
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <array>

using namespace std;

double get_walltime() // s
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double)(tp.tv_sec * 1e3 + tp.tv_usec * 1e-3);
}

const int BLOCK_DIM = 1024;
constexpr int BLOCK_DIM_WARP = 32;

__global__ void softmax_one_warp_one_row(float *input, float *output, int M, int N)
{
    int row = blockIdx.x;
    float val = -__FLT_MAX__;

    for (int i = threadIdx.x; i < N; i += BLOCK_DIM_WARP)
    {
        val = max(val, input[row * N + i]);
    }
    __syncthreads();
    for (int offset = BLOCK_DIM_WARP / 2; offset > 0; offset /= 2)
    {
        val = max(val, __shfl_down_sync(0xffffffff, val, offset));
    }
    __syncthreads();
    float globalMax = __shfl_sync(0xffffffff, val, 0);

    val = 0.0f;
    for (int i = threadIdx.x; i < N; i += BLOCK_DIM_WARP)
    {
        val += __expf(input[row * N + i] - globalMax);
    }
    __syncthreads();
    for (int offset = BLOCK_DIM_WARP / 2; offset > 0; offset /= 2)
    {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    __syncthreads();
    float globalSum = __shfl_sync(0xffffffff, val, 0);

    for (int i = threadIdx.x; i < N; i += BLOCK_DIM_WARP)
    {
        output[row * N + i] = __expf(input[row * N + i] - globalMax) / globalSum;
    }
}
__global__ void softmax_one_warp_one_row_float4(float *input, float *output, int M, int N)
{
    int row_id = blockIdx.x;
    int id_in_warp = threadIdx.x;
    int lane_id = id_in_warp % 32;

    float4 val4 = make_float4(-__FLT_MAX__, -__FLT_MAX__, -__FLT_MAX__, -__FLT_MAX__);
    for (int i = lane_id * 4; i < N; i += 32 * 4)
    {
        float4 data = *reinterpret_cast<float4*>(&input[row_id * N + i]);
        val4.x = max(val4.x, data.x);
        val4.y = max(val4.y, data.y);
        val4.z = max(val4.z, data.z);
        val4.w = max(val4.w, data.w);
    }

    float max_val = max(max(val4.x, val4.y), max(val4.z, val4.w));
    for (int offset = 16; offset > 0; offset /= 2)
    {
        max_val = max(max_val, __shfl_down_sync(0xffffffff, max_val, offset));
    }

    __shared__ float warpMax;
    if (lane_id == 0)
    {
        warpMax = max_val;
    }
    __syncthreads();

    float4 sum4 = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    for (int i = lane_id * 4; i < N; i += 32 * 4)
    {
        float4 data = *reinterpret_cast<float4*>(&input[row_id * N + i]);
        sum4.x += __expf(data.x - warpMax);
        sum4.y += __expf(data.y - warpMax);
        sum4.z += __expf(data.z - warpMax);
        sum4.w += __expf(data.w - warpMax);
    }

    float sum_val = sum4.x + sum4.y + sum4.z + sum4.w;
    for (int offset = 16; offset > 0; offset /= 2)
    {
        sum_val += __shfl_down_sync(0xffffffff, sum_val, offset);
    }

    __shared__ float warpSum;
    if (lane_id == 0)
    {
        warpSum = sum_val;
    }
    __syncthreads();

    for (int i = lane_id * 4; i < N; i += 32 * 4)
    {
        float4 data = *reinterpret_cast<float4*>(&input[row_id * N + i]);
        data.x = __expf(data.x - warpMax) / warpSum;
        data.y = __expf(data.y - warpMax) / warpSum;
        data.z = __expf(data.z - warpMax) / warpSum;
        data.w = __expf(data.w - warpMax) / warpSum;
        *reinterpret_cast<float4*>(&output[row_id * N + i]) = data;
    }
}

__global__ void softmax_one_warp_four_row(float *input, float *output, int M, int N)
{
    constexpr int size = 4;
    constexpr int group_size = BLOCK_DIM_WARP / size;
    int row_group_id = blockIdx.x;
    int warp_group_id = threadIdx.x / group_size;
    int id_in_warp = threadIdx.x % group_size;
    int gap = group_size;

    __shared__ float globalMax[size];
    __shared__ float globalSum[size];

    float val = -__FLT_MAX__;
    for (int i = id_in_warp; i < N; i += gap)
    {
        val = max(val, input[row_group_id * size * N + warp_group_id * N + i]);
    }
    for (int offset = gap / 2; offset > 0; offset /= 2)
    {
        val = max(val, __shfl_down_sync(0xff, val, offset, 8));
    }
    if (id_in_warp == 0)
    {
        globalMax[warp_group_id] = val;
    }
    __syncthreads();

    val = 0.0f;
    for (int i = id_in_warp; i < N; i += gap)
    {
        val += __expf(input[row_group_id * size * N + warp_group_id * N + i] - globalMax[warp_group_id]);
    }
    for (int offset = gap / 2; offset > 0; offset /= 2)
    {
        val += __shfl_down_sync(0xff, val, offset, 8);
    }
    if (id_in_warp == 0)
    {
        globalSum[warp_group_id] = val;
    }
    __syncthreads();

    for (int i = id_in_warp; i < N; i += gap)
    {
        output[row_group_id * 4 * N + warp_group_id * N + i] = __expf(input[row_group_id * 4 * N + warp_group_id * N + i] - globalMax[warp_group_id]) / globalSum[warp_group_id];
    }
}
__global__ void softmax_one_warp_four_row_float4(float *input, float *output, int M, int N)
{
    constexpr int size = 4;
    constexpr int group_size = BLOCK_DIM_WARP / size;
    int row_group_id = blockIdx.x;
    int warp_group_id = threadIdx.x / group_size;
    int id_in_warp = threadIdx.x % group_size;
    int gap = group_size;

    __shared__ float globalMax[size];
    __shared__ float globalSum[size];

    float4 val4 = make_float4(-__FLT_MAX__, -__FLT_MAX__, -__FLT_MAX__, -__FLT_MAX__);
    for (int i = id_in_warp * 4; i < N; i += gap * 4)
    {
        float4 data = *reinterpret_cast<float4 *>(&input[row_group_id * size * N + warp_group_id * N + i]);
        val4.x = max(val4.x, data.x);
        val4.y = max(val4.y, data.y);
        val4.z = max(val4.z, data.z);
        val4.w = max(val4.w, data.w);
    }
    float val = max(max(val4.x, val4.y), max(val4.z, val4.w));
    for (int offset = gap / 2; offset > 0; offset /= 2)
    {
        val = max(val, __shfl_down_sync(0xffffffff, val, offset, group_size));
    }
    if (id_in_warp == 0)
    {
        globalMax[warp_group_id] = val;
    }
    __syncthreads();

    float4 sum4 = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    for (int i = id_in_warp * 4; i < N; i += gap * 4)
    {
        float4 data = *reinterpret_cast<float4 *>(&input[row_group_id * size * N + warp_group_id * N + i]);
        sum4.x += __expf(data.x - globalMax[warp_group_id]);
        sum4.y += __expf(data.y - globalMax[warp_group_id]);
        sum4.z += __expf(data.z - globalMax[warp_group_id]);
        sum4.w += __expf(data.w - globalMax[warp_group_id]);
    }
    float sum = sum4.x + sum4.y + sum4.z + sum4.w;
    for (int offset = gap / 2; offset > 0; offset /= 2)
    {
        sum += __shfl_down_sync(0xffffffff, sum, offset, group_size);
    }
    if (id_in_warp == 0)
    {
        globalSum[warp_group_id] = sum;
    }
    __syncthreads();

    for (int i = id_in_warp * 4; i < N; i += gap * 4)
    {
        float4 data = *reinterpret_cast<float4 *>(&input[row_group_id * size * N + warp_group_id * N + i]);
        data.x = __expf(data.x - globalMax[warp_group_id]) / globalSum[warp_group_id];
        data.y = __expf(data.y - globalMax[warp_group_id]) / globalSum[warp_group_id];
        data.z = __expf(data.z - globalMax[warp_group_id]) / globalSum[warp_group_id];
        data.w = __expf(data.w - globalMax[warp_group_id]) / globalSum[warp_group_id];
        *reinterpret_cast<float4 *>(&output[row_group_id * size * N + warp_group_id * N + i]) = data;
    }
}

__global__ void softmax_one_warp_two_row_float4(float *input, float *output, int M, int N)
{
    constexpr int size = 2;
    constexpr int group_size = BLOCK_DIM_WARP / size;
    int row_group_id = blockIdx.x;
    int warp_group_id = threadIdx.x / group_size;
    int id_in_warp = threadIdx.x % group_size;
    int gap = group_size;

    __shared__ float globalMax[size];
    __shared__ float globalSum[size];

    float4 val4 = make_float4(-__FLT_MAX__, -__FLT_MAX__, -__FLT_MAX__, -__FLT_MAX__);
    for (int i = id_in_warp * 4; i < N; i += gap * 4)
    {
        float4 data = *reinterpret_cast<float4 *>(&input[row_group_id * size * N + warp_group_id * N + i]);
        val4.x = max(val4.x, data.x);
        val4.y = max(val4.y, data.y);
        val4.z = max(val4.z, data.z);
        val4.w = max(val4.w, data.w);
    }
    float val = max(max(val4.x, val4.y), max(val4.z, val4.w));
    for (int offset = gap / 2; offset > 0; offset /= 2)
    {
        val = max(val, __shfl_down_sync(0xffffffff, val, offset, group_size));
    }
    if (id_in_warp == 0)
    {
        globalMax[warp_group_id] = val;
    }
    __syncthreads();

    float4 sum4 = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    for (int i = id_in_warp * 4; i < N; i += gap * 4)
    {
        float4 data = *reinterpret_cast<float4 *>(&input[row_group_id * size * N + warp_group_id * N + i]);
        sum4.x += __expf(data.x - globalMax[warp_group_id]);
        sum4.y += __expf(data.y - globalMax[warp_group_id]);
        sum4.z += __expf(data.z - globalMax[warp_group_id]);
        sum4.w += __expf(data.w - globalMax[warp_group_id]);
    }
    float sum = sum4.x + sum4.y + sum4.z + sum4.w;
    for (int offset = gap / 2; offset > 0; offset /= 2)
    {
        sum += __shfl_down_sync(0xffffffff, sum, offset, group_size);
    }
    if (id_in_warp == 0)
    {
        globalSum[warp_group_id] = sum;
    }
    __syncthreads();

    for (int i = id_in_warp * 4; i < N; i += gap * 4)
    {
        float4 data = *reinterpret_cast<float4 *>(&input[row_group_id * size * N + warp_group_id * N + i]);
        data.x = __expf(data.x - globalMax[warp_group_id]) / globalSum[warp_group_id];
        data.y = __expf(data.y - globalMax[warp_group_id]) / globalSum[warp_group_id];
        data.z = __expf(data.z - globalMax[warp_group_id]) / globalSum[warp_group_id];
        data.w = __expf(data.w - globalMax[warp_group_id]) / globalSum[warp_group_id];
        *reinterpret_cast<float4 *>(&output[row_group_id * size * N + warp_group_id * N + i]) = data;
    }
}
__global__ void softmax_one_warp_two_row(float *input, float *output, int M, int N)
{
    constexpr int size = 2;
    constexpr int group_size = BLOCK_DIM_WARP / size;
    int row_group_id = blockIdx.x;
    int warp_group_id = threadIdx.x / group_size;
    int id_in_warp = threadIdx.x % group_size;
    int gap = group_size;

    __shared__ float globalMax[size];
    __shared__ float globalSum[size];

    float2 val2 = make_float2(-__FLT_MAX__, -__FLT_MAX__);
    for (int i = id_in_warp * 2; i < N; i += gap * 2)
    {
        float2 data = *reinterpret_cast<float2 *>(&input[row_group_id * size * N + warp_group_id * N + i]);
        val2.x = max(val2.x, data.x);
        val2.y = max(val2.y, data.y);
    }
    float val = max(val2.x, val2.y);
    for (int offset = gap / 2; offset > 0; offset /= 2)
    {
        val = max(val, __shfl_down_sync(0xffffffff, val, offset, group_size));
    }
    if (id_in_warp == 0)
    {
        globalMax[warp_group_id] = val;
    }
    __syncthreads();

    float2 sum2 = make_float2(0.0f, 0.0f);
    for (int i = id_in_warp * 2; i < N; i += gap * 2)
    {
        float2 data = *reinterpret_cast<float2 *>(&input[row_group_id * size * N + warp_group_id * N + i]);
        sum2.x += __expf(data.x - globalMax[warp_group_id]);
        sum2.y += __expf(data.y - globalMax[warp_group_id]);
    }
    float sum = sum2.x + sum2.y;
    for (int offset = gap / 2; offset > 0; offset /= 2)
    {
        sum += __shfl_down_sync(0xffffffff, sum, offset, group_size);
    }
    if (id_in_warp == 0)
    {
        globalSum[warp_group_id] = sum;
    }
    __syncthreads();

    for (int i = id_in_warp * 2; i < N; i += gap * 2)
    {
        float2 data = *reinterpret_cast<float2 *>(&input[row_group_id * size * N + warp_group_id * N + i]);
        data.x = __expf(data.x - globalMax[warp_group_id]) / globalSum[warp_group_id];
        data.y = __expf(data.y - globalMax[warp_group_id]) / globalSum[warp_group_id];
        *reinterpret_cast<float2 *>(&output[row_group_id * size * N + warp_group_id * N + i]) = data;
    }
}

__global__ void softmax(float *input, float *output, int M, int N)
{
    int row = blockIdx.x;
    __shared__ float tmp[BLOCK_DIM];
    __shared__ float globalMax;
    __shared__ float globalSum;

    float val = -__FLT_MAX__;
    for (int i = threadIdx.x; i < N; i += BLOCK_DIM)
    {
        val = max(val, input[row * N + i]);
    }
    tmp[threadIdx.x] = val;
    __syncthreads();

    for (int step = BLOCK_DIM / 2; step > 0; step /= 2)
    {
        if (threadIdx.x < step)
        {
            tmp[threadIdx.x] = max(tmp[threadIdx.x], tmp[threadIdx.x + step]);
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        globalMax = tmp[0];
    }
    __syncthreads();

    val = 0.0f;
    for (int i = threadIdx.x; i < N; i += BLOCK_DIM)
    {
        val += __expf(input[row * N + i] - globalMax);
    }
    tmp[threadIdx.x] = val;
    __syncthreads();

    for (int step = BLOCK_DIM / 2; step > 0; step /= 2)
    {
        if (threadIdx.x < step)
        {
            tmp[threadIdx.x] += tmp[threadIdx.x + step];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        globalSum = tmp[0];
    }
    __syncthreads();

    for (int i = threadIdx.x; i < N; i += BLOCK_DIM)
    {
        output[row * N + i] = __expf(input[row * N + i] - globalMax) * __fdividef(1.0F, globalSum);
    }
}

void cpu_softmax(float *cpu_input, float *cpu_output, int M, int N, float *input, float *output)
{
    double tot_ker_time = 0.0;
    double tot_time = 0.0;
    constexpr int times = 1001;

    int num_block = M;
    dim3 block_dim(BLOCK_DIM, 1, 1);
    dim3 grid_dim(num_block, 1, 1);
    for (int i = 0; i < times; i++)
    {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipDeviceSynchronize();
        double stt_tot_time = get_walltime();

        hipEventRecord(start, 0);

        softmax<<<grid_dim, block_dim>>>(input, output, M, N);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float ker_time = 0.0f;
        hipEventElapsedTime(&ker_time, start, stop);
        double end_tot_time = get_walltime();

        if (i > 0)
        {
            tot_ker_time += ker_time;
            tot_time += (end_tot_time - stt_tot_time);
        }

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    hipMemcpy(cpu_output, output, M * N * sizeof(float), hipMemcpyDeviceToHost);
    double avg_ker_time = tot_ker_time / (times - 1);
    double avg_tot_time = tot_time / (times - 1);
    printf("baseline: \n");
    printf("M = %d, N = %d \n", M, N);
    printf("average kernel time: %.4f ms, average total time: %.4f ms\n", avg_ker_time, avg_tot_time);
}

void cpu_softmax_one_warp_one_row(float *cpu_input, float *cpu_output, int M, int N, float *input, float *output)
{
    double tot_ker_time = 0.0;
    double tot_time = 0.0;
    constexpr int times = 1001;

    int num_block = M;
    dim3 block_dim(BLOCK_DIM_WARP, 1, 1);
    dim3 grid_dim(num_block, 1, 1);
    for (int i = 0; i < times; i++)
    {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipDeviceSynchronize();
        double stt_tot_time = get_walltime();

        hipEventRecord(start, 0);

        softmax_one_warp_one_row<<<grid_dim, block_dim>>>(input, output, M, N);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float ker_time = 0.0f;
        hipEventElapsedTime(&ker_time, start, stop);
        double end_tot_time = get_walltime();

        if (i > 0)
        {
            tot_ker_time += ker_time;
            tot_time += (end_tot_time - stt_tot_time);
        }

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    hipMemcpy(cpu_output, output, M * N * sizeof(float), hipMemcpyDeviceToHost);
    double avg_ker_time = tot_ker_time / (times - 1);
    double avg_tot_time = tot_time / (times - 1);
    printf("one warp one row: \n");
    printf("M = %d, N = %d \n", M, N);
    printf("average kernel time: %.4f ms, average total time: %.4f ms\n", avg_ker_time, avg_tot_time);
}
void cpu_softmax_one_warp_one_row_float4(float *cpu_input, float *cpu_output, int M, int N, float *input, float *output)
{
    double tot_ker_time = 0.0;
    double tot_time = 0.0;
    constexpr int times = 1001;

    int num_block = M;
    dim3 block_dim(BLOCK_DIM_WARP, 1, 1);
    dim3 grid_dim(num_block, 1, 1);
    for (int i = 0; i < times; i++)
    {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipDeviceSynchronize();
        double stt_tot_time = get_walltime();

        hipEventRecord(start, 0);

        softmax_one_warp_one_row_float4<<<grid_dim, block_dim>>>(input, output, M, N);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float ker_time = 0.0f;
        hipEventElapsedTime(&ker_time, start, stop);
        double end_tot_time = get_walltime();

        if (i > 0)
        {
            tot_ker_time += ker_time;
            tot_time += (end_tot_time - stt_tot_time);
        }

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    hipMemcpy(cpu_output, output, M * N * sizeof(float), hipMemcpyDeviceToHost);
    double avg_ker_time = tot_ker_time / (times - 1);
    double avg_tot_time = tot_time / (times - 1);
    printf("one warp one row float4 version: \n");
    printf("M = %d, N = %d \n", M, N);
    printf("average kernel time: %.4f ms, average total time: %.4f ms\n", avg_ker_time, avg_tot_time);
}
void cpu_softmax_one_warp_two_row(float *cpu_input, float *cpu_output, int M, int N, float *input, float *output)
{
    double tot_ker_time = 0.0;
    double total_time = 0.0;
    constexpr int times = 1001;

    int size = 2;
    int num_block = M / size;
    dim3 block_dim(BLOCK_DIM_WARP, 1, 1);
    dim3 grid_dim(num_block, 1, 1);
    for (int i = 0; i < times; i++)
    {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipDeviceSynchronize();
        double start_total = get_walltime();

        hipEventRecord(start, 0);

        softmax_one_warp_two_row<<<grid_dim, block_dim>>>(input, output, M, N);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float ker_time = 0.0f;
        hipEventElapsedTime(&ker_time, start, stop);
        double end_total = get_walltime();

        if (i > 0)
        {
            tot_ker_time += ker_time;
            total_time += (end_total - start_total);
        }

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    hipMemcpy(cpu_output, output, M * N * sizeof(float), hipMemcpyDeviceToHost);
    double avg_ker_time = tot_ker_time / (times - 1);
    double avg_tot_time = total_time / (times - 1);
    printf("one warp two rows: \n");
    printf("M = %d, N = %d \n", M, N);
    printf("average kernel time: %.4f ms, average total time: %.4f s\n", avg_ker_time, avg_tot_time);
}
void cpu_softmax_one_warp_two_row_float4(float *cpu_input, float *cpu_output, int M, int N, float *input, float *output)
{
    double tot_ker_time = 0.0;
    double total_time = 0.0;
    constexpr int times = 1001;

    int size = 2;
    int num_block = M / size;
    dim3 block_dim(BLOCK_DIM_WARP, 1, 1);
    dim3 grid_dim(num_block, 1, 1);
    for (int i = 0; i < times; i++)
    {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipDeviceSynchronize();
        double start_total = get_walltime();

        hipEventRecord(start, 0);

        softmax_one_warp_two_row_float4<<<grid_dim, block_dim>>>(input, output, M, N);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float ker_time = 0.0f;
        hipEventElapsedTime(&ker_time, start, stop);
        double end_total = get_walltime();

        if (i > 0)
        {
            tot_ker_time += ker_time;
            total_time += (end_total - start_total);
        }

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    hipMemcpy(cpu_output, output, M * N * sizeof(float), hipMemcpyDeviceToHost);
    double avg_ker_time = tot_ker_time / (times - 1);
    double avg_tot_time = total_time / (times - 1);
    printf("one warp two rows float4 version: \n");
    printf("M = %d, N = %d \n", M, N);
    printf("average kernel time: %.4f ms, average total time: %.4f s\n", avg_ker_time, avg_tot_time);
}

void cpu_softmax_one_warp_four_row(float *cpu_input, float *cpu_output, int M, int N, float *input, float *output)
{
    double tot_ker_time = 0.0;
    double tot_time = 0.0;
    constexpr int times = 1001;

    int size = 4;
    int num_block = M / size;
    dim3 block_dim(BLOCK_DIM_WARP, 1, 1);
    dim3 grid_dim(num_block, 1, 1);
    for (int i = 0; i < times; i++)
    {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipDeviceSynchronize();
        double stt_tot_time = get_walltime();

        hipEventRecord(start, 0);

        softmax_one_warp_four_row<<<grid_dim, block_dim>>>(input, output, M, N);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float ker_time = 0.0f;
        hipEventElapsedTime(&ker_time, start, stop);
        double end_tot_time = get_walltime();

        if (i > 0)
        {
            tot_ker_time += ker_time;
            tot_time += (end_tot_time - stt_tot_time);
        }

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    hipMemcpy(cpu_output, output, M * N * sizeof(float), hipMemcpyDeviceToHost);
    double avg_ker_time = tot_ker_time / (times - 1);
    double avg_tot_time = tot_time / (times - 1);
    printf("one warp four row: \n");
    printf("M = %d, N = %d \n", M, N);
    printf("average kernel time: %.4f ms, average total time: %.4f ms\n", avg_ker_time, avg_tot_time);
}
void cpu_softmax_one_warp_four_row_float4(float *cpu_input, float *cpu_output, int M, int N, float *input, float *output)
{
    double tot_ker_time = 0.0;
    double tot_time = 0.0;
    constexpr int times = 1001;

    int size = 4;
    int num_block = M / size;
    dim3 block_dim(BLOCK_DIM_WARP, 1, 1);
    dim3 grid_dim(num_block, 1, 1);
    for (int i = 0; i < times; i++)
    {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipDeviceSynchronize();
        double stt_tot_time = get_walltime();

        hipEventRecord(start, 0);

        softmax_one_warp_four_row_float4<<<grid_dim, block_dim>>>(input, output, M, N);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float ker_time = 0.0f;
        hipEventElapsedTime(&ker_time, start, stop);
        double end_tot_time = get_walltime();

        if (i > 0)
        {
            tot_ker_time += ker_time;
            tot_time += (end_tot_time - stt_tot_time);
        }

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    hipMemcpy(cpu_output, output, M * N * sizeof(float), hipMemcpyDeviceToHost);
    double avg_ker_time = tot_ker_time / (times - 1);
    double avg_tot_time = tot_time / (times - 1);
    printf("one warp four row float4 version: \n");
    printf("M = %d, N = %d \n", M, N);
    printf("average kernel time: %.4f ms, average total time: %.4f ms\n", avg_ker_time, avg_tot_time);
}

int main()
{
    double stt_tot_time = get_walltime();
    float *cpu_input, *cpu_output;
    float *input, *output;
    int M = 1024;
    int N = 1024;
    constexpr array<int, 3> Ms = {32, 1024, 2048};
    constexpr array<int, 3> Ns = {32, 1024, 2048};
    for (auto m : Ms)
    {
        for (auto n : Ns)
        {
            M = m;
            N = n;
            cpu_input = (float *)malloc(M * N * sizeof(float));
            cpu_output = (float *)malloc(M * N * sizeof(float));
            hipMalloc((void **)&input, M * N * sizeof(float));
            hipMalloc((void **)&output, M * N * sizeof(float));
            for (int i = 0; i < M * N; i++)
            {
                cpu_input[i] = i % 10;
            }
            hipMemcpy(input, cpu_input, M * N * sizeof(float), hipMemcpyHostToDevice);
            cpu_softmax(cpu_input, cpu_output, M, N, input, output);
            for (int i = 0; i < 10; i++)
            {
                printf("%.4e ", cpu_output[i]);
            }
            printf("\n\n");
            cpu_softmax_one_warp_one_row(cpu_input, cpu_output, M, N, input, output);
            for (int i = 0; i < 10; i++)
            {
                printf("%.4e ", cpu_output[i]);
            }
            printf("\n\n");
            cpu_softmax_one_warp_one_row_float4(cpu_input, cpu_output, M, N, input, output);
            for (int i = 0; i < 10; i++)
            {
                printf("%.4e ", cpu_output[i]);
            }
            printf("\n\n");
            cpu_softmax_one_warp_two_row(cpu_input, cpu_output, M, N, input, output);
            for (int i = 0; i < 10; i++)
            {
                printf("%.4e ", cpu_output[i]);
            }
            printf("\n\n");
            cpu_softmax_one_warp_two_row_float4(cpu_input, cpu_output, M, N, input, output);
            for (int i = 0; i < 10; i++)
            {
                printf("%.4e ", cpu_output[i]);
            }
            printf("\n\n");
            cpu_softmax_one_warp_four_row(cpu_input, cpu_output, M, N, input, output);
            for (int i = 0; i < 10; i++)
            {
                printf("%.4e ", cpu_output[i]);
            }
            printf("\n\n");
            cpu_softmax_one_warp_four_row_float4(cpu_input, cpu_output, M, N, input, output);
            for (int i = 0; i < 10; i++)
            {
                printf("%.4e ", cpu_output[i]);
            }
            printf("\n\n\n\n");
            hipFree(input);
            hipFree(output);
            free(cpu_input);
            free(cpu_output);
        }
    }
    double end_tot_time = get_walltime();
    double tot_time = end_tot_time - stt_tot_time;
    printf("\n\nthe total time of the whole program is: %.4f ms", tot_time);
    return 0;
}
